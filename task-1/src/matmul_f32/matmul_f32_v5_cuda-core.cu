#include "../xmx/fp32/fp32_v5.cuh"
#include "playground/matmul.hpp"
#include <cstdio>

namespace playground
{
// Implement the matmul function with DType=float16_t and Version=2
PLAYGROUND_MATMUL_DEC(float32_t, 5, M, N, K, A, B, C)
{
    // ......
    dim3 blocksize = {16, 32};
    unsigned int gridX = static_cast<unsigned int>((N + 128 - 1) / 128);
    unsigned int gridY = static_cast<unsigned int>((M + 128 - 1) / 128);
    dim3 gridsize = {gridX, gridY};
    gemm_v5<<<gridsize, blocksize>>>(M, N, K, A, B, C);

    // 检查核函数启动是否成功
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("核函数启动错误: %s\n", hipGetErrorString(err));
    }

    // 同步设备并检查执行期间的错误
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("核函数执行错误: %s\n", hipGetErrorString(err));
    }
}
}  // namespace playground