#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/library_types.h>

#include "../xmx/fp16/fp16_v17.cuh"
#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"
#include <cstdio>
namespace playground
{
PLAYGROUND_MATMUL_DEC(float16_t, 17, M, N, K, A, B, C)
{

    dim3 dimBlock(32, 2, 4);
    dim3 dimGrid(N / 128, M / 256);
    size_t smem_size = (128 * 32 + 256 * 32) * sizeof(float16_t) * 4;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error before kernel: %s\n", hipGetErrorString(err));
    }
    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_fp16_v17),
                         hipFuncAttributeMaxDynamicSharedMemorySize, 131072);
    gemm_fp16_v17<<<dimGrid, dimBlock, smem_size>>>(A, B, C, M, N, K);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error after kernel: %s\n", hipGetErrorString(err));
    }
}
}  // namespace playground
