#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/library_types.h>

#include "../xmx/fp16/fp16_v22.cuh"
#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"
#include <cstdio>
namespace playground
{
PLAYGROUND_MATMUL_DEC(float16_t, 22, M, N, K, A, B, C)
{

    dim3 dimBlock(32, 2, 4);
    dim3 dimGrid(N / 128, M / 256);
    size_t smem_size = max((128 * 32 + 256 * 32) * sizeof(float16_t) * 4,
                           (256 * 128) * sizeof(float16_t));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error before kernel: %s\n", hipGetErrorString(err));
    }
    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_fp16_v22),
                         hipFuncAttributeMaxDynamicSharedMemorySize, 131072);
    gemm_fp16_v22<<<dimGrid, dimBlock, smem_size>>>(A, B, C, M, N, K);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error after kernel: %s\n", hipGetErrorString(err));
    }
}
}  // namespace playground
